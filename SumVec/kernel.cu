#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const int* A, const int* B, int* C, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int n = 10000;  
    size_t size = n * sizeof(int);

    
    int* h_A = (int*)malloc(size);
    int* h_B = (int*)malloc(size);
    int* h_C = (int*)malloc(size);

   
    for (int i = 0; i < n; i++) {
        h_A[i] = 10;
        h_B[i] = 3;
    }

   
    int* d_A, * d_B, * d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, n);

    
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    
    std::cout << "Resultado: ";
    for (int i = 0; i < n; i++) {
        std::cout << h_C[i] << " ";
    }
    std::cout << std::endl;

    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
